#include "hip/hip_runtime.h"
#include <math.h>
#include <float.h>
// #include "quadrilinear4d_kernel.h"
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/numpy.h>
#include <torch/extension.h>
using namespace std;
#include <assert.h>
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>


#define CUDA_1D_KERNEL_LOOP(i, n)                            \
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
            i += blockDim.x * gridDim.x)

__device__ half fmod(half a, half b)
{
    half t = __habs(__hdiv(a,b));
    t = __hsub(t, hfloor(t));
    half c = __hmul(t, __habs(b));
    return (__hlt(a, __int2half_rd(0))) ? __hsub(__int2half_rd(0), c) : c;   /* if ( a < 0 ) c = 0-c */
}

__global__ void QuadriLinearForward(const int nthreads, const half* luts, const half* tri_index, const half* weight, const half* image1, const half* image2, const half* image3, const half* image4, half* output, const int luts_num, const int dim, const int shift, const half binsize, const int width, const int height, const int batch) {
        CUDA_1D_KERNEL_LOOP(index, nthreads) {
        
        int index_batch = floor(index / (width * height));
        int index_height = floor((index - (index_batch * width * height)) / width);
        int index_width = index - index_batch * width * height - index_height * width;
        // index = index_batch * width * height + index_height * width + index_width
        for (int index_channel = 0; index_channel < 3; index_channel += 1)
        {
            half a = image1[index_batch*3*height*width + index_channel*height*width + index_height*width + index_width];
            half b = image2[index_batch*3*height*width + index_channel*height*width + index_height*width + index_width];
            half c = image3[index_batch*3*height*width + index_channel*height*width + index_height*width + index_width];
            half d = image4[index_batch*3*height*width + index_channel*height*width + index_height*width + index_width];
            
            int a_id = __half2int_rd(hfloor(__hdiv(a, binsize)));
            int b_id = __half2int_rd(hfloor(__hdiv(b, binsize)));
            int c_id = __half2int_rd(hfloor(__hdiv(c, binsize)));
            int d_id = __half2int_rd(hfloor(__hdiv(d, binsize)));

            int a_d = __half2int_rd(fmod(a,binsize));
            int b_d = __half2int_rd(fmod(b,binsize));
            int c_d = __half2int_rd(fmod(c,binsize));
            int d_d = __half2int_rd(fmod(d,binsize));

            //TODO
	    int t = __half2int_rd(binsize);
            int tri_index_id = (a_d*t*t*t + b_d*t*t + c_d*t + d_d)*7;
            half sorted_a_d = __hdiv(tri_index[tri_index_id], binsize);
            half sorted_b_d = __hdiv(tri_index[tri_index_id+1], binsize);
            half sorted_c_d = __hdiv(tri_index[tri_index_id+2], binsize);
            half sorted_d_d = __hdiv(tri_index[tri_index_id+3], binsize);
            int index_o1 = __half2int_rd(tri_index[tri_index_id+4]);
            int index_o2 = __half2int_rd(tri_index[tri_index_id+5]);
            int index_o3 = __half2int_rd(tri_index[tri_index_id+6]);

            int index_o1_4 = index_o1 & 1;
            int index_o1_3 = (index_o1 >> 1) & 1;
            int index_o1_2 = (index_o1 >> 2) & 1;
            int index_o1_1 = (index_o1 >> 3) & 1;
            int index_o2_4 = index_o2 & 1;
            int index_o2_3 = (index_o2 >> 1) & 1;
            int index_o2_2 = (index_o2 >> 2) & 1;
            int index_o2_1 = (index_o2 >> 3) & 1;
            int index_o3_4 = index_o3 & 1;
            int index_o3_3 = (index_o3 >> 1) & 1;
            int index_o3_2 = (index_o3 >> 2) & 1;
            int index_o3_1 = (index_o3 >> 3) & 1;

            int id0 = (a_id * dim * dim * dim + b_id * dim * dim + c_id * dim + d_id)*shift*shift;
            int id1 = ((a_id + index_o1_1) * dim * dim * dim + (b_id + index_o1_2) * dim * dim + (c_id + index_o1_3) * dim + (d_id + index_o1_4))*shift*shift;
            int id2 = ((a_id + index_o2_1) * dim * dim * dim + (b_id + index_o2_2) * dim * dim + (c_id + index_o2_3) * dim + (d_id + index_o2_4))*shift*shift;
            int id3 = ((a_id + index_o3_1) * dim * dim * dim + (b_id + index_o3_2) * dim * dim + (c_id + index_o3_3) * dim + (d_id + index_o3_4))*shift*shift;
            int id4 = ((a_id + 1) * dim * dim * dim + (b_id + 1) * dim * dim + (c_id + 1) * dim + (d_id + 1))*shift*shift;

            half w0 = __hsub(__int2half_rd(1), sorted_a_d);
            half w1 = __hsub(sorted_a_d, sorted_b_d);
            half w2 = __hsub(sorted_b_d, sorted_c_d);
            half w3 = __hsub(sorted_c_d, sorted_d_d);
            half w4 = sorted_d_d;

            // 4x4 output pixel
            int lut_step = dim * dim * dim * dim * shift * shift;
            int output_index = (index_batch*3*height*width + index_channel*height*width + index_height*width + index_width)*shift*shift;
            // for each LUT base 
            for (int j = 0; j < luts_num; j += 1)
                // for 4x4 pixel
                for (int i = 0; i < shift*shift; i += 1)
                {   
                    half w = weight[index_batch*luts_num*height*width + j*height*width + index_height*width + index_width];
                    output[output_index+i] = __hadd(output[output_index+i],  __hmul(w, (__hadd(__hadd(__hadd(__hadd(__hmul(w0, luts[j*lut_step+id0+i]), __hmul(w1, luts[j*lut_step+id1+i])),  __hmul(w2, luts[j*lut_step+id2+i])), __hmul(w3, luts[j*lut_step+id3+i])), __hmul(w4, luts[j*lut_step+id4+i])))));
                }
        }
    }
}


int QuadriLinearForwardLaucher(const half* luts, const half* tri_index, const half* weight, const half* image1, const half* image2, const half* image3, const half* image4, half* output, const int luts_num, const int luts_dim, const int shift, const half binsize, const int width, const int height, const int batch) {
    const int kThreadsPerBlock = 512;
    const int output_size = height * width * batch;
    hipError_t err;


    QuadriLinearForward<<<(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0>>>(output_size, luts, tri_index, weight, image1, image2, image3, image4, output, luts_num,  luts_dim, shift, binsize, width, height, batch);

    err = hipGetLastError();
    if(hipSuccess != err) {
        fprintf( stderr, "cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
        exit( -1 );
    }

    return 1;
}

int quadrilinear4d_forward_cuda(torch::Tensor luts, torch::Tensor tri_index, torch::Tensor weight, torch::Tensor image1, torch::Tensor image2, torch::Tensor image3, torch::Tensor image4, torch::Tensor output,
                           int luts_num, int luts_dim, int shift, half binsize, int width, int height, int batch)
{
    // Grab the input tensor
    half * luts_flat = (half*)luts.data<float>();
    half * tri_index_flat = (half*)tri_index.data<float>();
    half * weight_flat = (half*)weight.data<float>();
    half * image_flat1 = (half*)image1.data<float>();
    half * image_flat2 = (half*)image2.data<float>();
    half * image_flat3 = (half*)image3.data<float>();
    half * image_flat4 = (half*)image4.data<float>();
    half * output_flat = (half*)output.data<float>();

    QuadriLinearForwardLaucher(luts_flat, tri_index_flat, weight_flat, image_flat1, image_flat2, image_flat3, image_flat4, output_flat, luts_num, luts_dim, shift, binsize, width, height, batch);

    return 1;
}


/*
__global__ void QuadriLinearBackward(const int nthreads, const half* luts, const half* tri_index, const half* weight, half* weight_grad, const half* image1, const half* image2, const half* image3, const half* image4, const half* output_grad, const int luts_num, const int dim, const int shift, const half binsize, const int width, const int height, const int batch) {
        CUDA_1D_KERNEL_LOOP(index, nthreads) {
        
        int index_batch = floor(index / (width * height));
        int index_height = floor((index - (index_batch * width * height)) / width);
        int index_width = index - index_batch * width * height - index_height * width;
        // index = index_batch * width * height + index_height * width + index_width
        for (int index_channel = 0; index_channel < 3; index_channel += 1)
        {
            half a = image1[index_batch*3*height*width + index_channel*height*width + index_height*width + index_width];
            half b = image2[index_batch*3*height*width + index_channel*height*width + index_height*width + index_width];
            half c = image3[index_batch*3*height*width + index_channel*height*width + index_height*width + index_width];
            half d = image4[index_batch*3*height*width + index_channel*height*width + index_height*width + index_width];
            
            int a_id = floor(a / binsize);
            int b_id = floor(b / binsize);
            int c_id = floor(c / binsize);
            int d_id = floor(d / binsize);

            int a_d = fmod(a,binsize);
            int b_d = fmod(b,binsize);
            int c_d = fmod(c,binsize);
            int d_d = fmod(d,binsize);

            //TODO
            int tri_index_id = (a_d*binsize*binsize*binsize + b_d*binsize*binsize + c_d*binsize + d_d)*7;
            half sorted_a_d = tri_index[tri_index_id] / binsize;
            half sorted_b_d = tri_index[tri_index_id+1] / binsize;
            half sorted_c_d = tri_index[tri_index_id+2] / binsize;
            half sorted_d_d = tri_index[tri_index_id+3] / binsize;
            int index_o1 = tri_index[tri_index_id+4];
            int index_o2 = tri_index[tri_index_id+5];
            int index_o3 = tri_index[tri_index_id+6];

            int index_o1_4 = index_o1 & 1;
            int index_o1_3 = (index_o1 >> 1) & 1;
            int index_o1_2 = (index_o1 >> 2) & 1;
            int index_o1_1 = (index_o1 >> 3) & 1;
            int index_o2_4 = index_o2 & 1;
            int index_o2_3 = (index_o2 >> 1) & 1;
            int index_o2_2 = (index_o2 >> 2) & 1;
            int index_o2_1 = (index_o2 >> 3) & 1;
            int index_o3_4 = index_o3 & 1;
            int index_o3_3 = (index_o3 >> 1) & 1;
            int index_o3_2 = (index_o3 >> 2) & 1;
            int index_o3_1 = (index_o3 >> 3) & 1;

            int id0 = (a_id * dim * dim * dim + b_id * dim * dim + c_id * dim + d_id)*shift*shift;
            int id1 = ((a_id + index_o1_1) * dim * dim * dim + (b_id + index_o1_2) * dim * dim + (c_id + index_o1_3) * dim + (d_id + index_o1_4))*shift*shift;
            int id2 = ((a_id + index_o2_1) * dim * dim * dim + (b_id + index_o2_2) * dim * dim + (c_id + index_o2_3) * dim + (d_id + index_o2_4))*shift*shift;
            int id3 = ((a_id + index_o3_1) * dim * dim * dim + (b_id + index_o3_2) * dim * dim + (c_id + index_o3_3) * dim + (d_id + index_o3_4))*shift*shift;
            int id4 = ((a_id + 1) * dim * dim * dim + (b_id + 1) * dim * dim + (c_id + 1) * dim + (d_id + 1))*shift*shift;

            half w0 = (1 - sorted_a_d);
            half w1 = (sorted_a_d - sorted_b_d);
            half w2 = (sorted_b_d - sorted_c_d);
            half w3 = (sorted_c_d - sorted_d_d);
            half w4 = sorted_d_d;

            // 4x4 output pixel
            int lut_step = dim * dim * dim * dim * shift * shift;
            int output_index = (index_batch*3*height*width + index_channel*height*width + index_height*width + index_width)*shift*shift;
            // for each LUT base 
            for (int j = 0; j < luts_num; j += 1)
                // for 4x4 pixel
                for (int i = 0; i < shift*shift; i += 1)
                {   
                    int w_index = index_batch*luts_num*height*width + j*height*width + index_height*width + index_width;
                    // atomicAdd(weight_grad + w_index, output_grad[output_index+i]*(w0 * luts[j*lut_step+id0+i] + w1 * luts[j*lut_step+id1+i] + w2 * luts[j*lut_step+id2+i] + w3 * luts[j*lut_step+id3+i] + w4 * luts[j*lut_step+id4+i]));
                    weight_grad[w_index] += output_grad[output_index+i]*(w0 * luts[j*lut_step+id0+i] + w1 * luts[j*lut_step+id1+i] + w2 * luts[j*lut_step+id2+i] + w3 * luts[j*lut_step+id3+i] + w4 * luts[j*lut_step+id4+i]);
                    // output[output_index+i] += w*(w0 * luts[j*lut_step+id0+i] + w1 * luts[j*lut_step+id1+i] + w2 * luts[j*lut_step+id2+i] + w3 * luts[j*lut_step+id3+i] + w4 * luts[j*lut_step+id4+i]);
                }
        }
    }
}
*/
int QuadriLinearBackwardLaucher(const half* luts, const half* tri_index, const half* weight, half* weight_grad, const half* image1, const half* image2, const half* image3, const half* image4, const half* output_grad, const int luts_num, const int luts_dim, const int shift, const half binsize, const int width, const int height, const int batch) {
/*    const int kThreadsPerBlock = 512;
    const int output_size = height * width * batch;
    hipError_t err;


    QuadriLinearBackward<<<(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0>>>(output_size, luts, tri_index, weight, weight_grad, image1, image2, image3, image4, output_grad, luts_num, luts_dim, shift, binsize, width, height, batch);

    err = hipGetLastError();
    if(hipSuccess != err) {
        fprintf( stderr, "cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
        exit( -1 );
    }*/

    return 1;
}

int quadrilinear4d_backward_cuda(torch::Tensor luts, torch::Tensor tri_index, torch::Tensor weight, torch::Tensor weight_grad, torch::Tensor image1, torch::Tensor image2, torch::Tensor image3, torch::Tensor image4, torch::Tensor output_grad,
                           int lut_num, int lut_dim, int shift, half binsize, int width, int height, int batch)
{
    // Grab the input tensor
    /*
    half * luts_flat = luts.data<half>();
    half * tri_index_flat = tri_index.data<half>();
    half * weight_flat = weight.data<half>();
    half * weight_grad_flat = weight_grad.data<half>();
    half * image_flat1 = image1.data<half>();
    half * image_flat2 = image2.data<half>();
    half * image_flat3 = image3.data<half>();
    half * image_flat4 = image4.data<half>();
    half * output_grad_flat = output_grad.data<half>();

    QuadriLinearBackwardLaucher(luts_flat, tri_index_flat, weight_flat, weight_grad_flat, image_flat1, image_flat2, image_flat3, image_flat4, output_grad_flat, lut_num, lut_dim, shift, binsize, width, height, batch);
*/
    return 1;
}
